#include "CudaUtils.h"

#include <hip/hip_runtime.h>

void* CudaUtils::AllocDeviceBuffer(size_t numBytes) {
  void*       d_buffer = nullptr;
  hipError_t status   = hipMalloc(&d_buffer, numBytes);
  if (status != hipSuccess) {
    std::cerr << "CUDA malloc failed: " << hipGetErrorString(status) << "\n";
    return nullptr;
  }
  return d_buffer;
}

void CudaUtils::FreeDeviceBuffer(void* d_buffer) {
  if (d_buffer) {
    hipError_t status = hipFree(d_buffer);
    if (status != hipSuccess) {
      std::cerr << "CUDA free failed: " << hipGetErrorString(status) << "\n";
    }
  }
}

bool CudaUtils::CopyToDevice(void* d_buffer, const void* h_buffer, size_t numBytes) {
  if (!d_buffer || !h_buffer) return false;

  hipError_t status = hipMemcpy(d_buffer, h_buffer, numBytes, hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    std::cerr << "CUDA memcpy HtoD failed: " << hipGetErrorString(status) << "\n";
    return false;
  }
  return true;
}

bool CudaUtils::CopyToHost(void* h_buffer, const void* d_buffer, size_t numBytes) {
  if (!h_buffer || !d_buffer) return false;

  hipError_t status = hipMemcpy(h_buffer, d_buffer, numBytes, hipMemcpyDeviceToHost);
  if (status != hipSuccess) {
    std::cerr << "CUDA memcpy DtoH failed: " << hipGetErrorString(status) << "\n";
    return false;
  }
  return true;
}

fbs::cuda::ipc::api::CudaIPCHandle CudaUtils::GetCudaMemoryHandle(void* d_ptr) {
  hipIpcMemHandle_t handle;
  hipError_t        status = hipIpcGetMemHandle(&handle, d_ptr);
  if (status != hipSuccess) {
    std::cerr << "Failed to get CUDA IPC handle: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipIpcGetMemHandle failed");
  }

  // Copy raw bytes into a vector
  //std::array<int64_t, 8> result{};
  //memcpy(result.data(), &handle, sizeof(handle));

  //flatbuffers::span<const uint8_t, 64> fb_span(
  //    reinterpret_cast<const uint8_t*>(&handle)
  //);

  auto result = fbs::cuda::ipc::api::CudaIPCHandle();
  return result;
}

void* CudaUtils::HandleToCudaMemory(const fbs::cuda::ipc::api::CudaIPCHandle& cuda_ipc_handle) {
  hipIpcMemHandle_t handle;
  static_assert(sizeof(handle) <= sizeof(cuda_ipc_handle), "Array size is too small for hipIpcMemHandle_t");
  memcpy(&handle, &cuda_ipc_handle, sizeof(handle));

  // Open the handle to get the device pointer
  void*       d_ptr  = nullptr;
  hipError_t status = hipIpcOpenMemHandle(&d_ptr, handle, hipIpcMemLazyEnablePeerAccess);
  if (status != hipSuccess) {
    std::cerr << "Failed to open CUDA IPC handle: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipIpcOpenMemHandle failed");
  }

  return d_ptr;
}
