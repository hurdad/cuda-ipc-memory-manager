#include <hip/hip_runtime.h>

#include <cstring>
#include <iostream>
#include <stdexcept>

#include "CudaUtils.h"

void CudaUtils::SetDevice(int device_id) {
  // Set the current CUDA device; throws on failure
  CUDA_CHECK(hipSetDevice(device_id));
  std::cout << "[CudaUtils] Set device to " << device_id << "\n";
}

void* CudaUtils::AllocDeviceBuffer(size_t numBytes, bool zeroInitialize) {
  void* d_buffer = nullptr;
  // Allocate device memory; throws on failure
  CUDA_CHECK(hipMalloc(&d_buffer, numBytes));

  // Optionally zero-initialize the buffer
  if (zeroInitialize) {
    CUDA_CHECK(hipMemset(d_buffer, 0, numBytes));
    std::cout << "[CudaUtils] Zero-initialized device buffer at " << d_buffer << " (" << numBytes << " bytes)\n";
  } else {
    std::cout << "[CudaUtils] Allocated device buffer at " << d_buffer << " (" << numBytes << " bytes)\n";
  }

  return d_buffer;
}

void CudaUtils::FreeDeviceBuffer(void* d_buffer) {
  if (!d_buffer) return;
  // Free device memory; throws on failure
  CUDA_CHECK(hipFree(d_buffer));
  std::cout << "[CudaUtils] Freed device buffer at " << d_buffer << "\n";
}

void CudaUtils::CopyToDevice(void* d_buffer, const void* h_buffer, size_t numBytes) {
  if (!d_buffer || !h_buffer) throw std::runtime_error("[CudaUtils] CopyToDevice failed: nullptr argument");
  // Copy from host to device; throws on failure
  CUDA_CHECK(hipMemcpy(d_buffer, h_buffer, numBytes, hipMemcpyHostToDevice));
  std::cout << "[CudaUtils] Copied " << numBytes << " bytes from host to device (" << d_buffer << ")\n";
}

void CudaUtils::CopyToHost(void* h_buffer, const void* d_buffer, size_t numBytes) {
  if (!h_buffer || !d_buffer) throw std::runtime_error("[CudaUtils] CopyToHost failed: nullptr argument");
  // Copy from device to host; throws on failure
  CUDA_CHECK(hipMemcpy(h_buffer, d_buffer, numBytes, hipMemcpyDeviceToHost));
  std::cout << "[CudaUtils] Copied " << numBytes << " bytes from device to host (" << d_buffer << ")\n";
}

const std::array<uint8_t, 64> CudaUtils::GetCudaMemoryHandle(void* d_ptr) {
  hipIpcMemHandle_t handle;
  // Create IPC handle for device memory; throws on failure
  CUDA_CHECK(hipIpcGetMemHandle(&handle, d_ptr));

  std::cout << "[CudaUtils] Created CUDA IPC handle for device pointer " << d_ptr << "\n";

  // Copy CUDA handle into std::array
  std::array<uint8_t, 64> handle_storage;
  std::memcpy(handle_storage.data(), &handle, sizeof(handle));
  return handle_storage;
}

void* CudaUtils::OpenHandleToCudaMemory(const std::array<uint8_t, 64>& cuda_ipc_handle) {
  hipIpcMemHandle_t handle;

  // Copy Flatbuffers data to CUDA handle
  std::memcpy(&handle, &cuda_ipc_handle, sizeof(handle));

  void* d_ptr = nullptr;
  // Open IPC handle; throws on failure
  CUDA_CHECK(hipIpcOpenMemHandle(&d_ptr, handle, hipIpcMemLazyEnablePeerAccess));

  std::cout << "[CudaUtils] Opened CUDA IPC handle, device pointer: " << d_ptr << "\n";
  return d_ptr;
}

void CudaUtils::CloseHandleToCudaMemory(void* d_ptr) {
  // Close IPC handle; throws on failure
  CUDA_CHECK(hipIpcCloseMemHandle(d_ptr));
  std::cout << "[CudaUtils] Closed CUDA IPC handle\n";
}

void CudaUtils::GetMemoryInfo(size_t* free, size_t* total) {
  // Get cuda mem info; throws on failure
  CUDA_CHECK(hipMemGetInfo(free, total));
}

int CudaUtils::GetDeviceIdFromUUID(const boost::uuids::uuid& gpu_uuid) {
  int device_count = 0;
  CUDA_CHECK(hipGetDeviceCount(&device_count));

  // search devices for uuid
  for (int i = 0; i < device_count; ++i) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));

    boost::uuids::uuid device_uuid;
    std::copy(std::begin(prop.uuid.bytes), std::end(prop.uuid.bytes), device_uuid.begin());

    if (device_uuid == gpu_uuid) {
      return i; // Found matching device
    }
  }

  throw std::runtime_error("No CUDA device found with the given GPU UUID : " + boost::uuids::to_string(gpu_uuid));
}