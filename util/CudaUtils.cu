#include "CudaUtils.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <cstring>


void CudaUtils::InitDevice(const int device_id) {
  hipError_t status = hipSetDevice(device_id);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipSetDevice failed: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipSetDevice failed");
  }

  std::cout << "[CudaUtils] Set device to " << device_id << "\n";
}

void* CudaUtils::AllocDeviceBuffer(size_t numBytes) {
  void*       d_buffer = nullptr;
  hipError_t status   = hipMalloc(&d_buffer, numBytes);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipMalloc failed for " << numBytes
        << " bytes: " << hipGetErrorString(status) << "\n";
    return nullptr;
  }

  std::cout << "[CudaUtils] Allocated device buffer at " << d_buffer
      << " (" << numBytes << " bytes)\n";
  return d_buffer;
}

void CudaUtils::FreeDeviceBuffer(void* d_buffer) {
  if (!d_buffer) return;

  hipError_t status = hipFree(d_buffer);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipFree failed for buffer " << d_buffer
        << ": " << hipGetErrorString(status) << "\n";
  } else {
    std::cout << "[CudaUtils] Freed device buffer at " << d_buffer << "\n";
  }
}

bool CudaUtils::CopyToDevice(void* d_buffer, const void* h_buffer, size_t numBytes) {
  if (!d_buffer || !h_buffer) {
    std::cerr << "[CudaUtils] CopyToDevice failed: nullptr argument\n";
    return false;
  }

  hipError_t status = hipMemcpy(d_buffer, h_buffer, numBytes, hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipMemcpy HtoD failed: " << hipGetErrorString(status) << "\n";
    return false;
  }

  std::cout << "[CudaUtils] Copied " << numBytes << " bytes from host to device ("
      << d_buffer << ")\n";
  return true;
}

bool CudaUtils::CopyToHost(void* h_buffer, const void* d_buffer, size_t numBytes) {
  if (!h_buffer || !d_buffer) {
    std::cerr << "[CudaUtils] CopyToHost failed: nullptr argument\n";
    return false;
  }

  hipError_t status = hipMemcpy(h_buffer, d_buffer, numBytes, hipMemcpyDeviceToHost);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipMemcpy DtoH failed: " << hipGetErrorString(status) << "\n";
    return false;
  }

  std::cout << "[CudaUtils] Copied " << numBytes << " bytes from device to host ("
      << d_buffer << ")\n";
  return true;
}

fbs::cuda::ipc::api::CudaIPCHandle CudaUtils::GetCudaMemoryHandle(void* d_ptr) {
  hipIpcMemHandle_t handle;
  hipError_t        status = hipIpcGetMemHandle(&handle, d_ptr);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipIpcGetMemHandle failed: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipIpcGetMemHandle failed");
  }

  std::cout << "[CudaUtils] Created CUDA IPC handle for device pointer " << d_ptr << "\n";

  // Convert CUDA handle to Flatbuffers handle
  flatbuffers::span<const uint8_t, 64> fb_span(
      reinterpret_cast<const uint8_t*>(&handle), sizeof(handle)
      );
  return fbs::cuda::ipc::api::CudaIPCHandle(fb_span);
}

void* CudaUtils::OpenHandleToCudaMemory(const fbs::cuda::ipc::api::CudaIPCHandle& cuda_ipc_handle) {
  hipIpcMemHandle_t handle;
  static_assert(sizeof(handle) <= sizeof(cuda_ipc_handle),
                "Array size is too small for hipIpcMemHandle_t");
  std::memcpy(&handle, &cuda_ipc_handle, sizeof(handle));

  void*       d_ptr  = nullptr;
  hipError_t status = hipIpcOpenMemHandle(&d_ptr, handle, hipIpcMemLazyEnablePeerAccess);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipIpcOpenMemHandle failed: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipIpcOpenMemHandle failed");
  }

  std::cout << "[CudaUtils] Opened CUDA IPC handle, device pointer: " << d_ptr << "\n";
  return d_ptr;
}

void CudaUtils::CloseHandleToCudaMemory(void* d_ptr) {
  // close ipc gpu memory
  hipError_t status = hipIpcCloseMemHandle(d_ptr);
  if (status != hipSuccess) {
    std::cerr << "[CudaUtils] hipIpcCloseMemHandle failed: " << hipGetErrorString(status) << "\n";
    throw std::runtime_error("hipIpcCloseMemHandle failed");
  }

  std::cout << "[CudaUtils] Close CUDA IPC handle \n";
}